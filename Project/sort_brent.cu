
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include<algorithm>
using namespace std;
#define BLOCKSIZE 256
#define DATASIZE 101

//INSERT CODE HERE---------------------------------
//Counting Sort
__global__ void countingData(int * pSource_d,int *offsetArray,int input_size){
	//Shared memory for saving data counts
	__shared__ int dataCounter[DATASIZE];

	int tx=threadIdx.x;
	int gx=blockIdx.x*blockDim.x + tx;
	
	//set initial value of array elements to 0
	if(tx<DATASIZE)
		dataCounter[tx]=0;
	__syncthreads();	
		
	if(gx<input_size)
		//atomically counts data
		atomicAdd(&(dataCounter[pSource_d[gx]]),1);
	__syncthreads();
	
	//add all shared memory values
	if(tx<DATASIZE)
		atomicAdd(&(offsetArray[tx]),dataCounter[tx]);
}

//Prefix Sum(Double-Buffered Kogge-Stone Parallel Scan Algorithm)
__global__ void prefixSum(int * pResult_d, int * offsetArray){
	__shared__ int T[DATASIZE];
	int stride=1;
	int index,i;
	int tx=threadIdx.x;
	while(stride<DATASIZE){
		index=(tx+1)*stride*2-1;
		if(index<DATASIZE)
			T[index]+=T[index-stride];
		stride*=2;

		__syncthreads();
	}
	stride=DATASIZE/2;
	while(stride>1){
		index=(tx+1)*stride*2-1;
		if(index<DATASIZE && (index+stride)<DATASIZE)
			T[index+stride]+=T[index];
		stride/=2;
		
		__syncthreads();
	}

	if(tx==0)
		for(i=0;i<T[tx];i++)
			pResult_d[i]=tx;
		
	else
		for(i=T[tx-1];i<T[tx];i++)
			pResult_d[i]=tx;
	offsetArray[tx]=T[tx];
	
}

void verify(int* src, int*result, int input_size){
	sort(src, src+input_size);
	long long match_cnt=0;
	for(int i=0; i<input_size;i++)
	{
		if(src[i]==result[i])
			match_cnt++;
	}

	if(match_cnt==input_size)
		printf("TEST PASSED\n\n");
	else
		printf("TEST FAILED\n\n");

}

void genData(int* ptr, unsigned int size) {
	while (size--) {
		*ptr++ = (int)(rand() % 101);
	}
}

int main(int argc, char* argv[]) {
	int* pSource = NULL;
	int* pResult = NULL;
	int* pOffset = NULL;
	int input_size=0;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	if (argc == 2)
		input_size=atoi(argv[1]);
	else
	{
    		printf("\n    Invalid input parameters!"
	   		"\n    Usage: ./sort <input_size>"
           		"\n");
        	exit(0);
	}

	//allocate host memory
	pSource=(int*)malloc(input_size*sizeof(int));
	pResult=(int*)malloc(input_size*sizeof(int));
	pOffset=(int*)malloc(DATASIZE*sizeof(int));
	// generate source data
	genData(pSource, input_size);
	
	
	// start timer
	hipEventRecord(start, 0);

	//INSERT CODE HERE--------------------
	//Device Memory
	int *pSource_d;
	int *pResult_d;
	int *offsetArray;

	//Device memory allocation
	hipMalloc((void**)&pSource_d,input_size*sizeof(int));
	hipMalloc((void**)&pResult_d,input_size*sizeof(int));
	hipMalloc((void**)&offsetArray,DATASIZE*sizeof(int));

	//Copy Host to Device
	hipMemcpy(pSource_d,pSource,input_size*sizeof(int),hipMemcpyHostToDevice);
	
	//launch kernel
	dim3 dimGrid(ceil((double)input_size/BLOCKSIZE),1,1);
    dim3 dimBlock(BLOCKSIZE,1,1);
    countingData<<< dimGrid, dimBlock>>>(pSource_d,offsetArray,input_size);
	hipDeviceSynchronize();

	prefixSum<<<1,DATASIZE>>>(pResult_d,offsetArray);
	hipDeviceSynchronize();
	//Copy Device to Host
	hipMemcpy(pResult,pResult_d,input_size*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(pOffset,offsetArray,DATASIZE*sizeof(int),hipMemcpyDeviceToHost);
	//Free Device Memory
	hipFree(pSource_d);
	hipFree(pResult_d);
	hipFree(offsetArray);

	// end timer
	float time;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("elapsed time = %f msec\n", time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("\n");
	for(int i=0;i<DATASIZE;i++)
		printf("%d ",pOffset[i]);
	printf("\n");
	printf("Verifying results..."); 
	fflush(stdout);
	verify(pSource, pResult, input_size);
	fflush(stdout);
}

