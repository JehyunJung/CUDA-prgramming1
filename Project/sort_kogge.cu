
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include<algorithm>
using namespace std;
#define BLOCKSIZE 256
#define DATASIZE 101

//INSERT CODE HERE---------------------------------
//Counting Sort
__global__ void countingData(int * pSource_d,int *offsetArray,int input_size){
	//Shared memory for saving data counts
	__shared__ int dataCounter[DATASIZE];

	int tx=threadIdx.x;
	int gx=blockIdx.x*blockDim.x + tx;
	
	//set initial value of array elements to 0
	if(tx<DATASIZE)
		dataCounter[tx]=0;
	__syncthreads();	
		
	if(gx<input_size)
		//atomically counts data
		atomicAdd(&(dataCounter[pSource_d[gx]]),1);
	__syncthreads();
	
	//add all shared memory values
	if(tx<DATASIZE)
		atomicAdd(&(offsetArray[tx]),dataCounter[tx]);
}

//Prefix Sum(Double-Buffered Kogge-Stone Parallel Scan Algorithm)
__global__ void prefixSum(int * pResult_d, int * offsetArray){
	__shared__ int source[2*DATASIZE];
	__shared__ int destination[2*DATASIZE];
	int tx=threadIdx.x;
	int temp;
	int stride=1;
	int index,i;

	source[tx]=0;
	destination[tx]=0;
	source[DATASIZE+tx]=offsetArray[tx];
	__syncthreads();

	while(1){
		index=2*DATASIZE-tx-1;
		destination[index]=source[index]+source[index-stride];
		__syncthreads();
		stride*=2;
		if(stride>DATASIZE)
			break;
		
		//Swap between arrays
		index=DATASIZE+tx;
		temp=source[index];
		source[index]=destination[index];
		destination[index]=temp;
	}

	for(i=destination[DATASIZE+tx-1];i<destination[DATASIZE+tx];i++){
		pResult_d[i]=tx;
	}
}

void verify(int* src, int*result, int input_size){
	sort(src, src+input_size);
	long long match_cnt=0;
	for(int i=0; i<input_size;i++)
	{
		if(src[i]==result[i])
			match_cnt++;
	}

	if(match_cnt==input_size)
		printf("TEST PASSED\n\n");
	else
		printf("TEST FAILED\n\n");

}

void genData(int* ptr, unsigned int size) {
	while (size--) {
		*ptr++ = (int)(rand() % 101);
	}
}

int main(int argc, char* argv[]) {
	int* pSource = NULL;
	int* pResult = NULL;
	int input_size=0;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	if (argc == 2)
		input_size=atoi(argv[1]);
	else
	{
    		printf("\n    Invalid input parameters!"
	   		"\n    Usage: ./sort <input_size>"
           		"\n");
        	exit(0);
	}

	//allocate host memory
	pSource=(int*)malloc(input_size*sizeof(int));
	pResult=(int*)malloc(input_size*sizeof(int));
	// generate source data
	genData(pSource, input_size);
	
	
	// start timer
	hipEventRecord(start, 0);

	//INSERT CODE HERE--------------------
	//Device Memory
	int *pSource_d;
	int *pResult_d;
	int *offsetArray;

	//Device memory allocation
	hipMalloc((void**)&pSource_d,input_size*sizeof(int));
	hipMalloc((void**)&pResult_d,input_size*sizeof(int));
	hipMalloc((void**)&offsetArray,DATASIZE*sizeof(int));

	//Copy Host to Device
	hipMemcpy(pSource_d,pSource,input_size*sizeof(int),hipMemcpyHostToDevice);
	
	//launch kernel
	dim3 dimGrid(ceil((double)input_size/BLOCKSIZE),1,1);
    dim3 dimBlock(BLOCKSIZE,1,1);
    countingData<<< dimGrid, dimBlock>>>(pSource_d,offsetArray,input_size);
	hipDeviceSynchronize();

	prefixSum<<<1,DATASIZE>>>(pResult_d,offsetArray);
	hipDeviceSynchronize();
	//Copy Device to Host
	hipMemcpy(pResult,pResult_d,input_size*sizeof(int),hipMemcpyDeviceToHost);

	//Free Device Memory
	hipFree(pSource_d);
	hipFree(pResult_d);
	hipFree(offsetArray);

	// end timer
	float time;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("elapsed time = %f msec\n", time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("Verifying results..."); 
	fflush(stdout);
	verify(pSource, pResult, input_size);
	fflush(stdout);
}
