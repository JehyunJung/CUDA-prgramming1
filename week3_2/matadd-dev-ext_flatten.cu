#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>
#include "common.h"
// Compute vector sum C = A+B
// Each thread performs one pairwise addition
__global__ void addKernel(const int* dev_a, const int* dev_b, int*dev_c, int n)
{
    int x = (blockIdx.x*blockDim.x)+threadIdx.x; //global index of x
    int y = (blockIdx.y*blockDim.y)+threadIdx.y; //global index of y
    int i = y * (blockDim.x) + x;   //actual index of i(1D array)
    //error handling
    if(i<n)
        dev_c[i] = dev_a[i] + dev_b[i];
}
int main(int argc, char*argv[]){
    int HEIGHT, WIDTH;
    int x,y;
    int *a,*b,*c;
    int *dev_a,*dev_b,*dev_c;
    int SIZE;
    
    if(argc<3){
        printf("Not inserted block size, Try again\n");
        exit(-1);
    }
    HEIGHT=atoi(argv[1]);
    WIDTH=atoi(argv[2]);
    SIZE=HEIGHT*WIDTH;

    a=(int*)malloc(sizeof(int*)*SIZE);
    b=(int*)malloc(sizeof(int*)*SIZE);
    c=(int*)malloc(sizeof(int*)*SIZE);


   for(y=0;y<HEIGHT;y++){
       for(x=0;x<WIDTH;x++){
           a[y*WIDTH+x]=y*10+x;
           b[y*WIDTH+x]=(y*10+x)*100;
        }
   }

    // Allocate device memory
    CUDA_CHECK( hipMalloc((void**)&dev_a, SIZE*sizeof(int)) );
    CUDA_CHECK( hipMalloc((void**)&dev_b, SIZE*sizeof(int)) );
    CUDA_CHECK( hipMalloc((void**)&dev_c, SIZE*sizeof(int)) );

    // Transfer A and B to device memory
    CUDA_CHECK(hipMemcpy(dev_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dev_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice));

    //dimension of thread block(x,y,z)
    dim3 dimBlock(WIDTH,HEIGHT,1);
    
    //configure grid ==> <<<number of thread blocks within grid, number of threads in each thread block>>> 
    addKernel<<<ceil(SIZE/256.0), 256>>>(dev_a, dev_b, dev_c,SIZE);
    
    // Transfer C from device to host
    CUDA_CHECK(hipMemcpy(c, dev_c, SIZE*sizeof(int), hipMemcpyDeviceToHost));

    // Free device memory for A, B, C
    CUDA_CHECK(hipFree(dev_a)); 
    CUDA_CHECK(hipFree(dev_b)); 
    CUDA_CHECK(hipFree(dev_c));

    for(y=0;y<HEIGHT;y++){
        for(x=0;x<WIDTH;x++)
            printf("%5d",c[y*WIDTH+x]);  
        printf("\n");
    }

    free(a);
    free(b);
    free(c);
    return 0;
}
